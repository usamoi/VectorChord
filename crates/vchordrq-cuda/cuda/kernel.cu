#include <assert.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#define QUOTE(x) vchordrq_assign_##x

extern "C" {

typedef float fp32;
typedef __half fp16;

typedef enum {
  vecf32_dot = 0,
  vecf32_l2s = 1,
  vecf16_dot = 2,
  vecf16_l2s = 3,
} op_t;

inline size_t size(op_t op) {
  switch (op) {
  case vecf32_dot:
    return 4;
  case vecf32_l2s:
    return 4;
  case vecf16_dot:
    return 2;
  case vecf16_l2s:
    return 2;
  default:
    return 0;
  }
}

typedef struct {
  op_t op;
  size_t d;
  size_t n;
  void *centroids;
} server_t;

typedef struct {
  hipStream_t stream;
  op_t op;
  size_t d;
  size_t n;
  void *centroids;
  size_t m;
  void *vectors;
  void *buffer;
  uint32_t *results;
} client_t;

server_t *QUOTE(server_alloc)(op_t op, size_t d, size_t n, void *centroids) {
  server_t *server = (server_t *)malloc(sizeof(server_t));
  server->op = op;
  server->d = d;
  server->n = n;
  server->centroids = NULL;
  if (hipMalloc(&server->centroids, n * d * size(op)) != hipSuccess) {
    free(server);
    return NULL;
  }
  if (hipMemcpy(server->centroids, centroids, n * d * size(op),
                 hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(server->centroids);
    free(server);
    return NULL;
  }
  return server;
}

void QUOTE(server_free)(server_t *server) {
  hipFree(server->centroids);
  free(server);
}

client_t *QUOTE(client_alloc)(op_t op, size_t d, size_t n, void *centroids,
                              size_t m) {
  client_t *client = (client_t *)malloc(sizeof(client_t));
  client->stream = NULL;
  client->op = op;
  client->d = d;
  client->n = n;
  client->centroids = NULL;
  client->m = m;
  client->vectors = NULL;
  client->buffer = NULL;
  client->results = NULL;
  client->centroids = centroids;
  if (hipStreamCreate(&client->stream) != hipSuccess) {
    free(client);
    return NULL;
  }
  if (hipMalloc(&client->vectors, m * d * size(op)) != hipSuccess) {
    hipStreamDestroy(client->stream);
    free(client);
    return NULL;
  }
  if (hipMalloc(&client->buffer, m * n * size(op)) != hipSuccess) {
    hipStreamDestroy(client->stream);
    hipFree(client->vectors);
    free(client);
    return NULL;
  }
  if (hipMalloc(&client->results, m * sizeof(uint32_t)) != hipSuccess) {
    hipStreamDestroy(client->stream);
    hipFree(client->buffer);
    hipFree(client->vectors);
    free(client);
    return NULL;
  }
  return client;
}

void QUOTE(client_free)(client_t *client) {
  hipStreamDestroy(client->stream);
  hipFree(client->results);
  hipFree(client->buffer);
  hipFree(client->vectors);
  free(client);
}

__global__ void QUOTE(kernel_0)(fp32 *centroids, size_t d, size_t n,
                                fp32 *vectors, uint32_t *results, size_t k) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= k) {
    return;
  }

  fp32 best_distance = INFINITY;
  size_t best_index = 0xffffffff;

  for (size_t index = 0; index < n; index++) {
    fp32 sum = 0.0f;
    for (size_t j = 0; j < d; j++) {
      fp32 x = vectors[idx * d + j];
      fp32 y = centroids[index * d + j];
      sum += x * y;
    }
    fp32 distance = -sum;
    if (distance < best_distance) {
      best_distance = distance;
      best_index = index;
    }
  }

  results[idx] = best_index;
}

__global__ void QUOTE(kernel_1)(fp32 *centroids, size_t d, size_t n,
                                fp32 *vectors, uint32_t *results, size_t k) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= k) {
    return;
  }

  fp32 best_distance = INFINITY;
  size_t best_index = 0xffffffff;

  for (size_t index = 0; index < n; index++) {
    fp32 sum = 0.0f;
    for (size_t j = 0; j < d; j++) {
      fp32 x = vectors[idx * d + j];
      fp32 y = centroids[index * d + j];
      fp32 diff = x - y;
      sum += diff * diff;
    }
    fp32 distance = sum;
    if (distance < best_distance) {
      best_distance = distance;
      best_index = index;
    }
  }

  results[idx] = best_index;
}

__global__ void QUOTE(kernel_2)(fp16 *centroids, size_t d, size_t n,
                                fp16 *vectors, uint32_t *results, size_t k) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= k) {
    return;
  }

  fp16 best_distance = INFINITY;
  size_t best_index = 0xffffffff;

  for (size_t index = 0; index < n; index++) {
    fp16 sum = 0.0f;
    for (size_t j = 0; j < d; j++) {
      fp16 x = vectors[idx * d + j];
      fp16 y = centroids[index * d + j];
      sum += x * y;
    }
    fp16 distance = -sum;
    if (distance < best_distance) {
      best_distance = distance;
      best_index = index;
    }
  }

  results[idx] = best_index;
}

__global__ void QUOTE(kernel_3)(fp16 *centroids, size_t d, size_t n,
                                fp16 *vectors, uint32_t *results, size_t k) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= k) {
    return;
  }

  fp16 best_distance = INFINITY;
  size_t best_index = 0xffffffff;

  for (size_t index = 0; index < n; index++) {
    fp16 sum = 0.0f;
    for (size_t j = 0; j < d; j++) {
      fp16 x = vectors[idx * d + j];
      fp16 y = centroids[index * d + j];
      fp16 diff = x - y;
      sum += diff * diff;
    }
    fp16 distance = sum;
    if (distance < best_distance) {
      best_distance = distance;
      best_index = index;
    }
  }

  results[idx] = best_index;
}

int QUOTE(client_query)(client_t *client, size_t k, void *vectors,
                        uint32_t *results) {
  assert(k <= client->m);

  int threads = 256;
  int blocks = (k + threads - 1) / threads;

  if (hipMemcpyAsync(client->vectors, vectors,
                      k * client->d * size(client->op), hipMemcpyHostToDevice,
                      client->stream) != hipSuccess) {
    return -1;
  }
  switch (client->op) {
  case vecf32_dot:
    QUOTE(kernel_0)<<<blocks, threads, 0, client->stream>>>(
        (fp32 *)client->centroids, client->d, client->n,
        (fp32 *)client->vectors, client->results, k);
    break;
  case vecf32_l2s:
    QUOTE(kernel_1)<<<blocks, threads, 0, client->stream>>>(
        (fp32 *)client->centroids, client->d, client->n,
        (fp32 *)client->vectors, client->results, k);
    break;
  case vecf16_dot:
    QUOTE(kernel_2)<<<blocks, threads, 0, client->stream>>>(
        (fp16 *)client->centroids, client->d, client->n,
        (fp16 *)client->vectors, client->results, k);
    break;
  case vecf16_l2s:
    QUOTE(kernel_3)<<<blocks, threads, 0, client->stream>>>(
        (fp16 *)client->centroids, client->d, client->n,
        (fp16 *)client->vectors, client->results, k);
    break;
  default:
    return -1;
  }
  if (hipMemcpyAsync(results, client->results, k * sizeof(uint32_t),
                      hipMemcpyDeviceToHost, client->stream) != hipSuccess) {
    return -1;
  }

  if (hipStreamSynchronize(client->stream) != hipSuccess) {
    return -1;
  }

  return 0;
}
}
